#include<pointcloud_cuda_mapping/grid_map_gpu.cuh>

GPU_GridMap::GPU_GridMap(int resolution,int width,int height){
    this->resolution=resolution;
    this->width=width;
    this->height=height;
    this->origin={0,0};
    hipMallocManaged(&(this->Map),width*height*sizeof(float));
};

GPU_GridMap::GPU_GridMap(int resolution,float width,float height){
    this->resolution=resolution;
    this->width=(int)(width/resolution);
    this->height=(int)(height/resolution);
    this->origin={0,0};
    hipMallocManaged(&(this->Map),this->width*this->height*sizeof(float));
};


GPU_GridMap::GPU_GridMap(int resolution,int width,int height,int origin_x, int origin_y){
    this->resolution=resolution;
    this->width=width;
    this->height=height;
    this->origin={origin_x,origin_y};
    hipMallocManaged(&(this->Map),width*height*sizeof(float));
};


GPU_GridMap::GPU_GridMap(int resolution,float width,float height, float origin_x, float origin_y){
    this->resolution=resolution;
    this->width=(int)(width/resolution);
    this->height=(int)(height/resolution);
    this->origin={origin_x/resolution,origin_y/resolution};
    hipMallocManaged(&(this->Map),this->width*this->height*sizeof(float));
};

GPU_GridMap::GPU_GridMap(int resolution,float width,float height, int origin_x, int origin_y){
    this->resolution=resolution;
    this->width=(int)(width/resolution);
    this->height=(int)(height/resolution);
    this->origin={origin_x/resolution,origin_y/resolution};
    this->origin={origin_x,origin_y};
    hipMallocManaged(&(this->Map),this->width*this->height*sizeof(float));
};

__device__ GPU_GridMap* GPU_GridMap::shift_to_gpu(){
    
}
__device__ int GPU_GridMap::indexOf(float x, float y){
    
}
