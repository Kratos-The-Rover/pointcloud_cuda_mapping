#include<pointcloud_cuda_mapping/grid_map_gpu.cuh>

GPU_GridMap::GPU_GridMap(int resolution,int width,int height){
    this->resolution=resolution;
    this->width=width;
    this->height=height;
    this->origin={0,0};
    hipMallocManaged(&(this->Map),width*height*sizeof(float));
};

GPU_GridMap::GPU_GridMap(int resolution,float width,float height){
    this->resolution=resolution;
    this->width=(int)(width/resolution);
    this->height=(int)(height/resolution);
    this->origin={0,0};
    hipMallocManaged(&(this->Map),this->width*this->height*sizeof(float));
};


GPU_GridMap::GPU_GridMap(int resolution,int width,int height,int origin_x, int origin_y){
    this->resolution=resolution;
    this->width=width;
    this->height=height;
    this->origin={origin_x,origin_y};
    hipMallocManaged(&(this->Map),width*height*sizeof(float));
};


GPU_GridMap::GPU_GridMap(int resolution,float width,float height, float origin_x, float origin_y){
    this->resolution=resolution;
    this->width=(int)(width/resolution);
    this->height=(int)(height/resolution);
    this->origin={origin_x/resolution,origin_y/resolution};
    hipMallocManaged(&(this->Map),this->width*this->height*sizeof(float));
};

GPU_GridMap::GPU_GridMap(int resolution,float width,float height, int origin_x, int origin_y){
    this->resolution=resolution;
    this->width=(int)(width/resolution);
    this->height=(int)(height/resolution);
    this->origin={origin_x/resolution,origin_y/resolution};
    this->origin={origin_x,origin_y};
    this->origin_x=origin_x;
    this->origin_y=origin_y;
    this->ptr_width= &this->width;
    this->ptr_height= &this->height;
    this->ptr_size= &this->map_size;
    this->ptr_resolution=&this->resolution;
    this->ptr_origin_x=&this->origin_x;
    this->ptr_origin_y=&this->origin_y;
    hipMallocManaged(&(this->Map),this->width*this->height*sizeof(float));
    hipMallocManaged(&(this->ptr_width),sizeof(float));
    hipMallocManaged(&(this->ptr_height),sizeof(float));
    hipMallocManaged(&(this->ptr_size),sizeof(float));

};

__device__ GPU_GridMap* GPU_GridMap::shift_to_gpu(){
    
}
int GPU_GridMap::indexOf(float x, float y){
    int x_map=(*(this->ptr_origin_x))+x/(*(this->ptr_resolution));
    int y_map=(*(this->ptr_origin_y))+y/(*(this->ptr_resolution));
    return x_map+this->width*y_map;
}


__device__ int GPU_GridMap::indexOfN(float x, float y){
    int x_map=this->origin_x+x/((this->resolution));
    int y_map=(this->origin_y)+y/(this->resolution);
    return x_map+this->width*y_map;
}
